#include "hip/hip_runtime.h"
template <typename Dtype>
__global__ void InitDistanceMetricKernel(
    const int nthreads, 
    Dtype* distanceMetric_data,
    const float factor,
    const int mW,
    const int mH,
    const int N) 
{
    CUDA_KERNEL_LOOP(n, nthreads) {
        const long q = n % mW; 
        const long p = (n / mW) % mH;
        const long j = (n / N) % mW;
        const long i = n / N / mW;
        const long u = i * mW + j;
        const long v = p * mW + q;
        if (u >= v) {
            *(distanceMetric_data + n) = expf(((i - p) * (i - p) + (j - q) * (j - q)) / (-2 * factor * factor));
            *(distanceMetric_data + v*N + u) = *(distanceMetric_data + n);
        }
    }
}

static int cuspn_SP_InitDistanceMetric(lua_State *L)
{
    THCState *state = getCutorchState(L);
    THCudaTensor *distanceMetric = (THCudaTensor*)luaT_checkudata(L, 2, "torch.CudaTensor");
    const float factor = lua_tonumber(L, 3);
    const long mW = lua_tonumber(L, 4);
    const long mH = lua_tonumber(L, 5);
    const long N = lua_tonumber(L, 6);
    // (mH*mW, mH*mW)
    float *distanceMetric_data = THCudaTensor_data(state, distanceMetric);

    const long count = N * N;
    InitDistanceMetricKernel <<< GET_BLOCKS(count), CUDA_NUM_THREADS, 0, THCState_getCurrentStream(state) >>>
        (count, distanceMetric_data, factor, mW, mH, N);
    THCudaCheck(hipGetLastError());

    return 1;
}

template <typename Dtype>
__global__ void InitTransferMatrixKernel(
    const int nthreads, 
    const Dtype* input_data,
    const Dtype* distanceMetric_data,
    Dtype* transferMatrix_data,
    const int nChannel,
    const int mW,
    const int mH,
    const int N) 
{
    CUDA_KERNEL_LOOP(n, nthreads) {
        const long q = n % mW; 
        const long p = (n / mW) % mH;
        const long j = (n / N) % mW;
        const long i = n / N / mW;
        const long u = i * mW + j;
        const long v = p * mW + q;

        if (i*j >= p*q) {
            long c;
            float sum = 0.0f;
            for (c = 0; c < nChannel; c++) {
                const float pntA = *(input_data + c * N + i * mW + j);
                const float pntB = *(input_data + c * N + p * mW + q);
                sum += (pntA - pntB) * (pntA - pntB);
            }
            *(transferMatrix_data + n) = sqrt(sum) * *(distanceMetric_data + n);
            *(transferMatrix_data + v*N + u) = *(transferMatrix_data + n);
        }
    }
}

template <typename Dtype>
__global__ void NormTransferMatrixKernel(
    const int nthreads, 
    Dtype* transferMatrix_data,
    const int N) 
{
    CUDA_KERNEL_LOOP(n, nthreads) {
        long c;
        float sum = 0.0f;
        for (c = 0; c < N; c++) {
            sum += *(transferMatrix_data + c * N + n);
        }
        for (c = 0; c < N; c++) {
            *(transferMatrix_data + c * N + n) /= sum;
        }
    }
}

template <typename Dtype>
__global__ void UpdateProposalKernel(
    const int nthreads, 
    const Dtype* src_data,
    const Dtype* diff_data,
    Dtype* dst_data,
    const float scale) 
{
    CUDA_KERNEL_LOOP(n, nthreads) {
        if (scale < 0) 
            *(dst_data + n) = *(src_data + n) + *(diff_data + n);
        else
            *(dst_data + n) = *(src_data + n) * scale;
    }
}

static int cuspn_SP_Generate(lua_State *L)
{
    THCState *state = getCutorchState(L);
    THCudaTensor *input = (THCudaTensor*)luaT_checkudata(L, 2, "torch.CudaTensor");
    THCudaTensor *distanceMetric = (THCudaTensor*)luaT_checkudata(L, 3, "torch.CudaTensor");
    THCudaTensor *transferMatrix = (THCudaTensor*)luaT_checkudata(L, 4, "torch.CudaTensor");
    THCudaTensor *proposal = (THCudaTensor*)luaT_checkudata(L, 5, "torch.CudaTensor");
    THCudaTensor *proposalBuffer = (THCudaTensor*)luaT_checkudata(L, 6, "torch.CudaTensor");
    THCUNN_assertSameGPU(state, 5, input, distanceMetric, transferMatrix, proposal, proposalBuffer);
    const long nBatch = lua_tonumber(L, 7);
    const long nChannel = lua_tonumber(L, 8);
    const long mW = lua_tonumber(L, 9);
    const long mH = lua_tonumber(L, 10);
    const long N = lua_tonumber(L, 11);
    const float tolerance = lua_tonumber(L, 12);
    const long maxIteration = lua_tonumber(L, 13);
    const long nEntry = nChannel * N;

    // (nBatch, nChannel, mH, mW)
    float *input_data = THCudaTensor_data(state, input);
    // (mH*mW, mH*mW)
    float *distanceMetric_data = THCudaTensor_data(state, distanceMetric);
    // (mH*mW, mH*mW)
    float *transferMatrix_data = THCudaTensor_data(state, transferMatrix);
    // (nBatch, mH, mW)
    float *proposal_data = THCudaTensor_data(state, proposal);
    // (mH, mW)
    float *proposalBuffer_data = THCudaTensor_data(state, proposalBuffer);

    const float avg = 1.0f / N;
    float sumOver;
    long count;
    long i, j;

    THCudaTensor_fill(state, proposal, avg);
    for (i = 0; i < nBatch; i++) {
        /* init transfer matrix */
        count = N * N;
        InitTransferMatrixKernel <<< GET_BLOCKS(count), CUDA_NUM_THREADS, 0, THCState_getCurrentStream(state) >>>
            (count, input_data + i * nEntry, distanceMetric_data, transferMatrix_data, nChannel, mW, mH, N);
        THCudaCheck(hipGetLastError());
        count = N;
        NormTransferMatrixKernel <<< GET_BLOCKS(count), CUDA_NUM_THREADS, 0, THCState_getCurrentStream(state) >>>
            (count, transferMatrix_data, N);
        THCudaCheck(hipGetLastError());

        /* generate soft proposal for each sample */
        // init buffer
        THCudaTensor_fill(state, proposalBuffer, avg);
        for (j = 0; j < maxIteration; j++) {
            // calculate diffs
            THCudaBlas_Sgemv(
                state,
                't',
                N,
                N,
                1.0f,
                transferMatrix_data,
                N,
                proposal_data,
                1,
                -1.0f,
                proposalBuffer_data,
                1
            );
            float normDiff = THCudaTensor_normall(state, proposalBuffer, 2);
            if (THCudaTensor_normall(state, proposalBuffer, 2) < tolerance) break;
            // add diffs
            UpdateProposalKernel <<< GET_BLOCKS(count), CUDA_NUM_THREADS, 0, THCState_getCurrentStream(state) >>>
                (count, proposal_data, proposalBuffer_data, proposalBuffer_data, -1.0f);
            THCudaCheck(hipGetLastError());

            sumOver = THCudaTensor_sumall(state, proposalBuffer);
            if (sumOver < 0) break;
            // update proposal
            UpdateProposalKernel <<< GET_BLOCKS(count), CUDA_NUM_THREADS, 0, THCState_getCurrentStream(state) >>>
                (count, proposalBuffer_data, proposal_data, proposal_data, 1.0f / sumOver);
            THCudaCheck(hipGetLastError());
        }
        proposal_data += N;
    }

    return 1;
}

template <typename Dtype>
__global__ void CoupleKernel(
    const int nthreads, 
    const Dtype* input_data,
    const Dtype* proposal_data,
    Dtype* output_data,
    const int nBatch,
    const int nChannel,
    const int N) 
{
    CUDA_KERNEL_LOOP(n, nthreads) {
        const long k = n % N;
        const long i = n / N / nChannel;
        *(output_data + n) = *(input_data + n) * *(proposal_data + i * N + k);
    }
}

static int cuspn_SP_Couple(lua_State *L)
{
    THCState *state = getCutorchState(L);
    THCudaTensor *input = (THCudaTensor*)luaT_checkudata(L, 2, "torch.CudaTensor");
    THCudaTensor *proposal = (THCudaTensor*)luaT_checkudata(L, 3, "torch.CudaTensor");
    THCudaTensor *output = (THCudaTensor*)luaT_checkudata(L, 4, "torch.CudaTensor");
    THCUNN_assertSameGPU(state, 3, input, proposal, output);
    const long nBatch = lua_tonumber(L, 5);
    const long nChannel = lua_tonumber(L, 6);
    const long N = lua_tonumber(L, 7);
    // (nBatch, nChannel, mH, mW)
    float *input_data = THCudaTensor_data(state, input);
    // (nBatch, mH, mW)
    float *proposal_data = THCudaTensor_data(state, proposal);
    // (nBatch, nChannel, mH, mW)
    float *output_data = THCudaTensor_data(state, output);

    const long count = nBatch * nChannel * N;
    CoupleKernel <<< GET_BLOCKS(count), CUDA_NUM_THREADS, 0, THCState_getCurrentStream(state) >>>
        (count, input_data, proposal_data, output_data, nBatch, nChannel, N);
    THCudaCheck(hipGetLastError());

    return 1;
}

static const struct luaL_Reg cuspn_SP__ [] = {
    {"SP_InitDistanceMetric", cuspn_SP_InitDistanceMetric},
    {"SP_Generate", cuspn_SP_Generate},
    {"SP_Couple", cuspn_SP_Couple},
    {NULL, NULL}
};

static void cuspn_SP_init(lua_State *L)
{
    luaT_pushmetatable(L, "torch.CudaTensor");
    luaT_registeratname(L, cuspn_SP__, "spn");
    lua_pop(L,1);
}
